#include "hip/hip_runtime.h"
/*
 * CPE 458
 * Ray Tracer/Sound Tracer
 * Professor Christopher Lupo
 * Bryan Ching(bcching), Matt Crussell(macrusse)
 */

#include <stdio.h>
#include <stdlib.h>
#include <cfloat>
#include <time.h>
#include "glm/glm.hpp"
#include <math.h>
#include <algorithm>
#include "Image.h"
#include "types.h"
#include "cudaRayTrace.h"

#include "fmod/inc/fmod.h"
#include "fmod/inc/fmod_errors.h"
#include "fmod/wincompat.h"

#define X_SIZE 1024 
#define Y_SIZE 128
#define BLOCK_SIZE 1024

Camera * camera, *cam_d;
PointLight *light, *l_d;
Plane * planes, *p_d;
Sphere * spheres, *s_d;
float * output_dist_d;
Point * output_vec_d;
float theta, stheta;
int lastx, lasty;
float *reduced_dist_d, *final_dist_d;
Point *reduced_vec_d, *final_vec_d;
int reductDim = X_SIZE*Y_SIZE/1024;


FMOD_SYSTEM     *asystem;
FMOD_SOUND      *sound1, *sound2, *sound3;
FMOD_CHANNEL    *channel1 = 0, *channel2 = 0, *channel3 = 0;
FMOD_RESULT      result;
int              key;
int              listenerflag = TRUE;
FMOD_VECTOR      listenerpos  = { 0.0f, 0.0f, 0.0f }; 
unsigned int     version;

Camera* CameraInit();
PointLight* LightInit();
Sphere* CreateSpheres();
Plane* CreatePlanes();

__host__ __device__ Point CreatePoint(float x, float y, float z);
__host__ __device__ color_t CreateColor(float r, float g, float b);

__global__ void CUDARayTrace(Camera * cam, Plane * f, PointLight *l, Sphere * s, uchar4 * position);
__global__ void computeAudio(int ear_dir,int forward, Point * o_vec3, float * o_distance,  Camera * cam,Plane * planes, Sphere * spheres);
__global__ void reduce(float *g_idata, Point *g_ivec, float *g_odata, Point * g_ovec, unsigned int n);

__device__ color_t RayTrace(Ray r, Sphere* s, Plane* f, PointLight* l);
__device__ color_t SphereShading(int sNdx, Ray r, Point p, Sphere* sphereList, PointLight* l);
__device__ color_t Shading(Ray r, Point p, Point normalVector, PointLight* l, color_t diffuse, color_t ambient, color_t specular); 
__device__ float SphereRayIntersection(Sphere* s, Ray r);
__device__ float PlaneRayIntersection(Plane* s, Ray r);
__device__ float findDistance(Ray myRay, Camera * cam, Plane * planes, Sphere * spheres);


/* 
 *  Handles CUDA errors, taking from provided sample code on clupo site
 */
static void HandleError( hipError_t err, const char * file, int line)
{
  if(err !=hipSuccess){
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

/*
 * Sets up Video/Audio Scene information
 */
extern "C" void setup_scene()
{
  HANDLE_ERROR(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
  
  //SCENE SETUP
  camera = CameraInit();
  light = LightInit();
  spheres = CreateSpheres();
  planes = CreatePlanes(); 

  //FMOD SOUND SETUP
  FMOD_System_Create(&asystem);
  FMOD_System_Init(asystem, 10, FMOD_INIT_NORMAL, NULL);
  FMOD_System_CreateSound(asystem, "808-clap.wav", FMOD_SOFTWARE | FMOD_3D, 0, &sound1);
  FMOD_Sound_SetMode(sound1, FMOD_LOOP_NORMAL);
  
  FMOD_System_PlaySound(asystem, FMOD_CHANNEL_FREE, sound1, TRUE, &channel1);
  FMOD_Channel_SetPaused(channel1, FALSE);

  //CUDA MALLOCS
  HANDLE_ERROR( hipMalloc((void**)&cam_d, sizeof(Camera)) );
  HANDLE_ERROR( hipMalloc(&p_d, sizeof(Plane)*NUM_PLANES) );
  HANDLE_ERROR( hipMalloc(&l_d, sizeof(PointLight)) );
  HANDLE_ERROR( hipMalloc(&s_d,  sizeof(Sphere)*NUM_SPHERES) );
  HANDLE_ERROR( hipMalloc(&output_dist_d, sizeof(float) * WINDOW_WIDTH * WINDOW_HEIGHT) );
  HANDLE_ERROR( hipMalloc(&output_vec_d, sizeof(Point) * WINDOW_WIDTH * WINDOW_HEIGHT) );
  HANDLE_ERROR( hipMalloc(&reduced_dist_d, sizeof(float) * reductDim) );
  HANDLE_ERROR( hipMalloc(&final_dist_d, sizeof(float)));
  HANDLE_ERROR( hipMalloc(&reduced_vec_d, sizeof(Point) * reductDim) );
  HANDLE_ERROR( hipMalloc(&final_vec_d, sizeof(Point)));

  //CUDA MEMCPYS
  HANDLE_ERROR( hipMemcpy(l_d, light, sizeof(PointLight), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(cam_d, camera,sizeof(Camera), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(p_d, planes,sizeof(Plane)*NUM_PLANES, hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );


  //INIT VALUES
  theta = 0;
  stheta = 0;
  lastx = -1;
  lasty = -1;
}

/*
 * Controls Perspective movement of camera
 */
extern "C" void ijklMove(int y, int x)
{
  float sin_theta_x, cos_theta_x, sin_theta_y,cos_theta_y;

  if(lastx == -1)
  {
    lastx = x;
    lasty = y;
    return;
  }
  if(x - lastx > 2 )
    camera->theta_x-= .025;
  else if(x - lastx < -2)	
    camera->theta_x+= .025;
  if(y - lasty > 2)
    camera->theta_y+=.025;
  else if(y- lasty < -2)	
    camera->theta_y-=.025;

  lastx = x;
  lasty = y;

  sin_theta_x = sin(camera->theta_x);
  sin_theta_y = sin(camera->theta_y);
  cos_theta_x = cos(camera->theta_x);
  cos_theta_y = cos(camera->theta_y);

  camera->lookAt = glm::normalize(CreatePoint(sin_theta_y ,sin_theta_x , -1*cos_theta_x*cos_theta_y));
  camera->lookRight = glm::normalize(CreatePoint(cos_theta_y , 0 , sin_theta_y));
  camera->lookUp = glm::normalize(CreatePoint(0,cos_theta_x, sin_theta_x));
}
/*
 * Controls directional movement of camera location
 */
extern "C" void wasdMove(unsigned char key)
{
  Point set;
  set.x = 1;
  set.y = 0;
  set.z = 1;

  Point tempAt = glm::normalize(camera->lookAt * set);
  Point tempRight = glm::normalize(camera->lookRight * set);

  Point move;
  switch(key){
    case('w'):
      move = 10.f * tempAt;
      break; 
    case('s'):
      move = -10.f *tempAt;
      break;
    case('a'):
      move = -10.f * tempRight;
      break;
    case('d'):
      move = 10.f * tempRight;
      break;
  }
  camera->eye += move;
}
/*
 * Control Misc Keyboard Commands of Virtual Room
 */
extern "C" void misc(unsigned char key)
{
  Point center;
  switch(key){
    case('q'):
      {
        camera = CameraInit();
        break;
      }
    case('r'):
      {
        spheres = CreateSpheres();
        HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
        break;
      }
    case('-'):
      {
        for(int i = 0; i < NUM_SPHERES-1; i++)
          spheres[i].radius = glm::max(0.f, spheres[i].radius-1);
        HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
        break;
      }
    case('='):
      {
        for(int i = 0; i < NUM_SPHERES-1; i++)
          spheres[i].radius = glm::min(1000.f, spheres[i].radius+1);
        HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
        break;
      }
    case('o'):
      {
        Point center = *new Point(0,0,-2400);
        center = *new Point(0,0,-2400);
        for(int i = 0; i < NUM_SPHERES; i++)
        {
          Point c_dir = glm::normalize(spheres[i].center - center);
          Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
          spheres[i].center += 5.f*move_dir;
          spheres[i].center -= 5.f*c_dir;

        }
        HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
        break;
      }
    case('p'):
      {
        center = *new Point(0,0,-2400);
        for(int i = 0; i < NUM_SPHERES; i++)
        {
          Point c_dir = glm::normalize(spheres[i].center - center);
          Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
          spheres[i].center -= 10.f*move_dir;
          spheres[i].center += 10.f*c_dir;

        }
        HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
        break;
      }
    case('['):
      {
        center = camera->eye;
        for(int i = 0; i < NUM_SPHERES-1; i++)
        {
          Point c_dir = glm::normalize(spheres[i].center - center);
          Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
          spheres[i].center += 10.f*move_dir;
        }
        HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
        break;
      }

    case(']'):
      {
        center = camera->eye;
        for(int i = 0; i < NUM_SPHERES-1; i++)
        {
          Point c_dir = glm::normalize(spheres[i].center - center);
          Point move_dir = glm::cross(c_dir, *new Point(0,1,0));
          spheres[i].center -= 10.f*move_dir;

        }
        HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );
        break;
      }
    case('9'):
      {

        for(int i = 0; i < NUM_PLANES; i++) {
          planes[i].ambient.r = glm::max(planes[i].ambient.r - .05f, 0.f);
          planes[i].ambient.g = glm::max(planes[i].ambient.g - .05f, 0.f);
          planes[i].ambient.b = glm::max(planes[i].ambient.b - .05f, 0.f);
          planes[i].diffuse.r = glm::max(planes[i].diffuse.r - .05f, 0.f);
          planes[i].diffuse.g = glm::max(planes[i].diffuse.g - .05f, 0.f);
          planes[i].diffuse.b = glm::max(planes[i].diffuse.b - .05f, 0.f);
        }

        HANDLE_ERROR( hipMemcpy(p_d, planes,sizeof(Plane)*NUM_PLANES, hipMemcpyHostToDevice) );
        break;
      }
    case('0'):
      {
        for(int i = 0; i < NUM_PLANES; i++) {
          planes[i].ambient.r = glm::min(planes[i].ambient.r + .05f, 1.f);
          planes[i].ambient.g = glm::min(planes[i].ambient.g + .05f, 1.f);
          planes[i].ambient.b = glm::min(planes[i].ambient.b + .05f, 1.f);
          planes[i].diffuse.r = glm::min(planes[i].diffuse.r + .05f, 1.f);
          planes[i].diffuse.g = glm::min(planes[i].diffuse.g + .05f, 1.f);
          planes[i].diffuse.b = glm::min(planes[i].diffuse.b + .05f, 1.f);
        }

        HANDLE_ERROR( hipMemcpy(p_d, planes,sizeof(Plane)*NUM_PLANES, hipMemcpyHostToDevice) );
        break;
      }
  }
}
/*
 * Launches Audio kernel for 4 quadrants, saves minimum distance and sets sound source to that point
 */
extern "C" void launch_audio_kernel()
{
  dim3 gridSize(X_SIZE/16, Y_SIZE/16);
  dim3 blockSize(16,16);

  float dist;
  Point vec, min_vec;
  float temp;

  //QUADRANT 1
  computeAudio<<<gridSize, blockSize>>>(1,1, output_vec_d, output_dist_d, cam_d, p_d, s_d);  
  hipDeviceSynchronize();
  reduce<<<reductDim, 1024>>>(output_dist_d, output_vec_d,reduced_dist_d, reduced_vec_d, X_SIZE*Y_SIZE);
  hipDeviceSynchronize();
  reduce<<<1,reductDim>>>(reduced_dist_d, reduced_vec_d, final_dist_d, final_vec_d, reductDim);

  HANDLE_ERROR( hipMemcpy(&dist, final_dist_d, sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR( hipMemcpy(&vec, final_vec_d, sizeof(Point), hipMemcpyDeviceToHost));
  min_vec = vec * dist; 
  temp = dist;

  //QUADRANT 2
  computeAudio<<<gridSize, blockSize>>>(1,-1, output_vec_d, output_dist_d, cam_d, p_d, s_d);  
  hipDeviceSynchronize();
  reduce<<<reductDim, 1024>>>(output_dist_d, output_vec_d,reduced_dist_d, reduced_vec_d, X_SIZE*Y_SIZE);
  hipDeviceSynchronize();
  reduce<<<1,reductDim>>>(reduced_dist_d, reduced_vec_d, final_dist_d, final_vec_d, reductDim);

  HANDLE_ERROR( hipMemcpy(&dist, final_dist_d, sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR( hipMemcpy(&vec, final_vec_d, sizeof(Point), hipMemcpyDeviceToHost));
  
  if(dist < temp)
  {
    min_vec = vec * dist; 
    temp = dist;
  }

  //QUADRANT 3
  computeAudio<<<gridSize, blockSize>>>(-1,1, output_vec_d, output_dist_d, cam_d, p_d, s_d);  
  hipDeviceSynchronize();
  reduce<<<reductDim, 1024>>>(output_dist_d, output_vec_d,reduced_dist_d, reduced_vec_d, X_SIZE*Y_SIZE);
  hipDeviceSynchronize();
  reduce<<<1,reductDim>>>(reduced_dist_d, reduced_vec_d, final_dist_d, final_vec_d, reductDim);

  HANDLE_ERROR( hipMemcpy(&dist, final_dist_d, sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR( hipMemcpy(&vec, final_vec_d, sizeof(Point), hipMemcpyDeviceToHost));
  
  if(dist < temp)
  {
    min_vec = vec * dist; 
    temp = dist;
  }

  //QUADRANT 4
  computeAudio<<<gridSize, blockSize>>>(-1,-1, output_vec_d, output_dist_d, cam_d, p_d, s_d);  
  hipDeviceSynchronize();
  reduce<<<reductDim, 1024>>>(output_dist_d, output_vec_d,reduced_dist_d, reduced_vec_d, X_SIZE*Y_SIZE);
  hipDeviceSynchronize();
  reduce<<<1,reductDim>>>(reduced_dist_d, reduced_vec_d, final_dist_d, final_vec_d, reductDim);
  
  HANDLE_ERROR( hipMemcpy(&dist, final_dist_d, sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR( hipMemcpy(&vec, final_vec_d, sizeof(Point), hipMemcpyDeviceToHost));
  
  if(dist < temp)
  {
    min_vec = vec * dist; 
    temp = dist;
  }

  //Scale Down Distances and Account for coord differences between FMOD and Ray Tracer
  min_vec /= 750;
  min_vec *= -1; 

  if(temp < 1000000)
  {

    FMOD_Channel_Set3DAttributes(channel2, (FMOD_VECTOR *) &min_vec, NULL);
    FMOD_Channel_SetMute(channel2, FALSE);
  }
  else
    FMOD_Channel_SetMute(channel2, TRUE);

}
/*
 * Launches Both Video and Audio Ray Tracing from GLM overhead
 */
extern "C" void launch_kernel(uchar4* pos, unsigned int image_width, 
    unsigned int image_height, float time)
{
  //Lighting Effect
  light->position.x -=  sin(theta += .01);
  if(theta == 360)
    theta = 0;
  spheres[NUM_SPHERES-1].center=light->position;
  HANDLE_ERROR( hipMemcpy(l_d, light, sizeof(PointLight), hipMemcpyHostToDevice) );

  //Update Camera information for Cam and FMOD
  HANDLE_ERROR( hipMemcpy(cam_d, camera,sizeof(Camera), hipMemcpyHostToDevice) );
  FMOD_VECTOR vel = {0.0f, 0.0f, 0.0f};
  Point temp = camera->eye;
  temp/=750;
  FMOD_System_Set3DListenerAttributes(asystem, 0, (FMOD_VECTOR *) &(temp), &vel,(FMOD_VECTOR *) &(camera->lookAt),(FMOD_VECTOR *) &(camera->lookUp));
  FMOD_System_Update(asystem);

  HANDLE_ERROR( hipMemcpy(s_d, spheres,sizeof(Sphere)*NUM_SPHERES, hipMemcpyHostToDevice) );



  // The Video Kernel Call
  dim3 gridSize((WINDOW_WIDTH+15)/16, (WINDOW_HEIGHT+15)/16);
  dim3 blockSize(16,16);
  CUDARayTrace<<< gridSize, blockSize  >>>(cam_d, p_d, l_d, s_d, pos);
  hipDeviceSynchronize();
 
  //Launch Audio Kernel
  launch_audio_kernel();

} 

/*
 * Initializes camera at point (X,Y,Z)
 */
Camera* CameraInit() {

  Camera* c = new Camera();

  c->eye = CreatePoint(0, 100, 0);//(X,Y,Z)
  c->lookAt = CreatePoint(0, 0, SCREEN_DISTANCE);
  c->lookUp = CreatePoint(0, 1, 0);
  c->lookRight = CreatePoint(1, 0, 0);
  c->theta_x = 0;
  c->theta_y = 0;
  return c;
}

/*
 * Initializes light at hardcoded (X,Y,Z) coordinates
 */
PointLight* LightInit() {
  PointLight* l = new PointLight();

  l->ambient = CreateColor(0.2, 0.2, 0.2);
  l->diffuse = CreateColor(0.6, 0.6, 0.6);
  l->specular = CreateColor(0.4, 0.4, 0.4);

  l->position = CreatePoint(0, 400, -750);

  return l;
}

/*
 * Creates a point, for GLM Point has been defined as vec3
 */
__host__  __device__ Point CreatePoint(float x, float y, float z) {
  Point p;

  p.x = x;
  p.y = y;
  p.z = z;

  return p;
}

/*
 * Creates a color_t type color based on input values
 */
__host__ __device__ color_t CreateColor(float r, float g, float b) {
  color_t c;

  c.r = r;
  c.g = g;
  c.b = b;
  c.f = 1.0;

  return c;
}

/*
 * Creates NUM_SPHERES # of Spheres, with randomly chosen values on color, location, and size
 */
Sphere* CreateSpheres() {
  Sphere* spheres = new Sphere[NUM_SPHERES]();
  float randr, randg, randb;
  int num = 0;
  while (num < NUM_SPHERES-1) {
    randr = (rand()%1000) /1000.f ;
    randg = (rand()%1000) /1000.f ;
    randb = (rand()%1000) /1000.f ;
    spheres[num].radius = 50. - rand() % 30;
    spheres[num].center = CreatePoint(600 - rand() % 1200,
        100,
        //    700 - rand() % 1100,
        500 - rand() %2500);
    spheres[num].ambient = CreateColor(randr, randg, randb);
    spheres[num].diffuse = CreateColor(randr, randg, randb);
    spheres[num].specular = CreateColor(randr, randg, randb);
    num++;
  }
  spheres[NUM_SPHERES -2].ambient = CreateColor(300,300,240);

  spheres[NUM_SPHERES-1].radius=30;
  spheres[NUM_SPHERES-1].center=light->position;
  spheres[NUM_SPHERES-1].ambient=CreateColor(300,300,240);
  spheres[NUM_SPHERES-1].diffuse=CreateColor(1,1,.8);
  spheres[NUM_SPHERES-1].specular=CreateColor(1,1,.8);

  return spheres;
}

/*
 * Creates NUM_PLANES NUMBER OF PLANES, CURRENTLY THIS IS HARDCODED
 */
Plane* CreatePlanes() {
  Plane* planes = new Plane[NUM_PLANES]();
  planes[0].normal = CreatePoint(0,1,0);
  planes[0].center = CreatePoint(0,-100,0);
  planes[0].ambient = planes[0].diffuse = planes[0].specular = CreateColor(1,1,1);

  planes[1].normal = CreatePoint(0,-1,0) ;
  planes[1].center = CreatePoint(0,400,0);
  planes[1].ambient = planes[1].diffuse = planes[1].specular = CreateColor(1,1,1);

  planes[2].normal = CreatePoint(0,0, 1) ;
  planes[2].center = CreatePoint(0,0,-1000);
  planes[2].ambient = planes[2].diffuse = planes[2].specular = CreateColor(1,1,1);

  planes[3].normal = CreatePoint(1,0,0) ;
  planes[3].center = CreatePoint(-600,0,0);
  planes[3].ambient = planes[3].diffuse = planes[3].specular = CreateColor(1,1,1);

  planes[4].normal = CreatePoint(-1,0,0) ;
  planes[4].center = CreatePoint(600,0, 0);
  planes[4].ambient = planes[4].diffuse = planes[4].specular = CreateColor(1,1,1);

  planes[5].normal = CreatePoint(0,0,-1) ;
  planes[5].center = CreatePoint(0,0, 500);
  planes[5].ambient = planes[5].diffuse = planes[5].specular = CreateColor(1,1,1);

  return planes;
}

/*
   Refactored, taken from http://www.nvidia.com/content/cudazone/cuda_sdk/Data-Parallel_Algorithms.html
   This version uses n/2 threads --
   it performs the first level of reduction when reading from global memory
 */
__global__ void reduce(float *g_idata,Point *g_ivec, float *g_odata, Point * g_ovec, unsigned int n)
{

  __shared__ float sdata[BLOCK_SIZE];
  __shared__ Point svec[BLOCK_SIZE];
  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

  float temp;

  //Perform first reduction moving to shared memory
  sdata[tid] = (i < n) ? g_idata[i] : FLT_MAX;
  if(i < n)
    svec[tid] = g_ivec[i];
 
  if (i + blockDim.x < n && (temp = g_idata[i+blockDim.x]) < sdata[tid])
  {
    sdata[tid] = temp;
    svec[tid] = g_ivec[i+blockDim.x];
  }
  __syncthreads();

  // do reduction in shared mem
  for(unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s && (temp = sdata[tid+s]) < sdata[tid])
    {
      sdata[tid] = temp;
      svec[tid] = svec[tid+s];
    }
    __syncthreads();
  }

  // write result for this block to global mem 
  if (tid == 0) 
  {
    g_odata[blockIdx.x] = sdata[0];
    g_ovec[blockIdx.x] = svec[0];
  }
}

/*
 * Computes Audio on Per-Quadrant basis. Values written to output vectors to be reduced later
 */
__global__ void computeAudio(int ear_dir, int forward, Point * o_vec3, float * o_distance,  Camera * cam,Plane * planes, Sphere * spheres)
{
  Ray myRay;

  int row = blockIdx.y*blockDim.y + threadIdx.y;
  int col = blockIdx.x*blockDim.x + threadIdx.x;
  int index = row * blockDim.x *gridDim.x + col;

  float tanVal = tan(FOV/2);
  float rvaly = tanVal - (2 * tanVal / Y_SIZE) * row;

  float rad = (float) col / X_SIZE * 3.141592 / 2.f;
  float rvalz = glm::sin(rad);
  float rvalx = glm::cos(rad);

  /*Ray direction info*/
  myRay.origin = cam->eye;
  myRay.direction = cam->lookRight * rvalx;
  myRay.direction += (rvaly * cam->lookUp);
  myRay.direction += rvalz * cam->lookAt;
  myRay.direction.x *= ear_dir;
  myRay.direction.z *= forward;
  myRay.direction = glm::normalize(myRay.direction);

  o_distance[index] = findDistance(myRay, cam, planes, spheres);
  o_vec3[index] = myRay.direction;
}
/*
 * Calculates the distance to speaker from Ray origin. FLT_MAX returned on no intersection.
 */
__device__ float findDistance(Ray myRay, Camera * cam, Plane * planes, Sphere * spheres)
{
  float total_distance = 0;
  Ray currentRay = myRay;
  int i, closestSphere, closestPlane;
  float smallest, t;

  for(int j = 0; j < 3; j++)//Loop for 2 reflections
  {
    i = 0;
    closestSphere = -1; 
    closestPlane = -1;
    smallest = 0;
    //FIND CLOSEST SPHERE ALONG RAY R
    while (i < NUM_SPHERES-1) {
      t = SphereRayIntersection(spheres + i, currentRay);

      if (t > 0 && (closestSphere < 0 || t < smallest)) {
        smallest = t;
        closestSphere = i;
      }
      i++;
    }
    i=0;
    while (i < NUM_PLANES) {
      t = PlaneRayIntersection(planes + i, currentRay);
      if (t > 0 && ( (closestSphere < 0 && closestPlane < 0) || t < smallest)) {//POSSIBLE LOGIC FIX CLOSESTSPHERE >1
        smallest = t;
        closestSphere = -1;
        closestPlane = i;
      }
      i++;
    } 
    if(smallest == 0)//N0 INTERSECTIONS
      return FLT_MAX;
    total_distance += smallest;

    if(closestSphere == NUM_SPHERES-2)//The Speaker(Hit)
      return total_distance;

    //Energy lost from wall
    total_distance+= 1000; 
    
    //Calculate New Ray
    currentRay.origin = currentRay.direction * smallest + currentRay.origin;
    if(closestPlane != -1)
    {
      currentRay.direction = -glm::reflect(-glm::normalize(currentRay.direction), glm::normalize(planes[closestPlane].normal));

    }
    else if(closestSphere < NUM_SPHERES-2)
    {
      currentRay.direction = -glm::reflect(-glm::normalize(currentRay.direction), glm::normalize(currentRay.origin - spheres[closestSphere].center));

    }
    currentRay.direction = glm::normalize(currentRay.direction);
  }

  //No intersection with speaker
  return FLT_MAX;
}

/*
 * CUDA global function which performs ray tracing. Responsible for initializing and writing to output vector
 */
__global__ void CUDARayTrace(Camera * cam,Plane * f,PointLight * l, Sphere * s, uchar4 * pos)
{
  float tanVal = tan(FOV/2);

  //CALCULATE ABSOLUTE ROW,COL
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  color_t returnColor;
  Ray r;

  //BOUNDARY CHECK
  if(row >= WINDOW_HEIGHT || col >= WINDOW_WIDTH)
    return;

  float rvaly = tanVal - (2 * tanVal / WINDOW_HEIGHT) * row;
  float rvalx = -1 * WINDOW_WIDTH / WINDOW_HEIGHT * tanVal + (2 * tanVal / WINDOW_HEIGHT) * col;
  //INIT RAY VALUES
  r.origin = cam->eye;
  r.direction = cam->lookAt;
  r.direction += (rvalx * cam->lookRight);
  r.direction += (rvaly * cam->lookUp);
  r.direction = glm::normalize(r.direction);

  //RAY TRACE
  returnColor = RayTrace(r, s, f, l);

  //CALC OUTPUT INDEX
  int index = row *WINDOW_WIDTH + col;

  //PLACE DATA IN INDEX
  pos[index].x = 0xFF * returnColor.r;
  pos[index].y = 0xFF * returnColor.g;
  pos[index].z = 0xFF * returnColor.b;
  pos[index].w = 0xFF * returnColor.f;

}

/*
 * Performs Ray tracing over all spheres for any ray r
 */
__device__ color_t RayTrace(Ray r, Sphere* s, Plane* f, PointLight* l) {
  color_t color = CreateColor(0, 0, 0); 
  float t, smallest;
  int i = 0, closestSphere = -1, closestPlane = -1,  inShadow = false;
  Point normalVector;
  //FIND CLOSEST SPHERE ALONG RAY R
  while (i < NUM_SPHERES) {
    t = SphereRayIntersection(s + i, r);

    if (t > 0 && (closestSphere < 0 || t < smallest)) {
      smallest = t;
      closestSphere = i;
    }
    i++;
  }
  i=0;
  while (i < NUM_PLANES) {
    t = PlaneRayIntersection(f + i, r);
    if (t > 0 && ( (closestSphere < 0 && closestPlane < 0) || t < smallest)) {//POSSIBLE LOGIC FIX CLOSESTSPHERE >1
      smallest = t;
      closestSphere = -1;
      closestPlane = i;
    }
    i++;
  }

  //SETUP FOR SHADOW CALCULATIONS
  i = 0;
  Ray shadowRay;

  //r.direction += r.origin;//Smallest needs to be calculated differently
  shadowRay.origin = CreatePoint(r.direction.x * smallest, r.direction.y * smallest, r.direction.z * smallest);
  shadowRay.origin += r.origin;
  shadowRay.direction = l->position - shadowRay.origin;

  //DETERMINE IF SPHERE IS BLOCKING RAY FROM LIGHT TO SPHERE
  if(closestSphere > -1 || closestPlane > -1)
  {
    while (i <NUM_SPHERES-1 && !inShadow){ 
      t = SphereRayIntersection(s + i, shadowRay);
      if(i != closestSphere && t < 1 && t > 0){
        inShadow = true;
      }
      i++;
    }
    i = 0;
    while(i < NUM_PLANES && !inShadow){
      t = PlaneRayIntersection(f + i, shadowRay);
      if(i != closestPlane && t < 1 && t > 0){
        inShadow = true;
      }
      i++;
    }
  }

  if(closestPlane > -1 && !inShadow)
  {
    //plane closer than sphere
    return Shading(r, shadowRay.origin, f[closestPlane].normal, l, f[closestPlane].diffuse,
        f[closestPlane].ambient,f[closestPlane].specular);
  }
  if(closestPlane > -1)
  {
    color.r = l->ambient.r * f[closestPlane].ambient.r;
    color.g = l->ambient.g * f[closestPlane].ambient.g;
    color.b = l->ambient.b * f[closestPlane].ambient.b;
    return color;
  }

  //IF SHADOWED, ONLY SHOW AMBIENT LIGHTING
  if(closestSphere > -1 && !inShadow)
  {

    normalVector = glm::normalize(shadowRay.origin-(s[closestSphere].center));
    return Shading(r, shadowRay.origin, normalVector, l, s[closestSphere].diffuse,
        s[closestSphere].ambient,s[closestSphere].specular);
  }
  if(closestSphere > -1)
  {
    color.r = l->ambient.r * s[closestSphere].ambient.r;
    color.g = l->ambient.g * s[closestSphere].ambient.g;
    color.b = l->ambient.b * s[closestSphere].ambient.b;
  }
  return color;
}

/*
 * Determines distance of intersection of Ray with Plane, -1 returned if no intersection
 */
__device__ float PlaneRayIntersection(Plane *p, Ray r)
{
  float t;
  float denominator = glm::dot(r.direction,p->normal);
  if(denominator!=0)
  {
    t = (glm::dot(p->center-r.origin,p->normal)) / denominator;
    if (t>1000000)
      return -1;
    return t;
  }
  else
  {
    return -1;
  }
}
/*
 * Determines distance of intersection of Ray with Sphere, -1 returned if no intersection
 * http://sci.tuomastonteri.fi/programming/sse/example3
 */
__device__ float SphereRayIntersection(Sphere* s, Ray r) {
  float a, b, c, d, t1, t2;

  a = glm::dot((r.direction), (r.direction));

  b = glm::dot((r.origin)-(s->center),(r.direction));
  c = glm::dot((s->center),(s->center)) +glm::dot(r.origin,r.origin) -2.0f*glm::dot(r.origin, s->center)
    - (s->radius * s->radius);
  d = (b * b) - (a * c);

  if (d >= 0) {

    t1 = (-1 * b - sqrt(d)) / (a);
    t2 = (-1 * b + sqrt(d)) / (a);

    if (t2 > t1 && t1 > 0) {
      return t1;

    } else if (t2 > 0) {
      return t2;

    }
  }
  return -1;
}
/*
 * Determines Ambient, Diffuse, and Specular lighting on the plane
 */ 
__device__ color_t Shading(Ray r, Point p, Point normalVector,
    PointLight* l, color_t diffuse, color_t ambient, color_t specular) {
  color_t a, d, s, total;
  float NdotL, RdotV;
  Point viewVector, lightVector, reflectVector;

  viewVector = glm::normalize((r.origin)-p);

  lightVector = glm::normalize((l->position) -p);

  NdotL = glm::dot(lightVector, normalVector);
  reflectVector = (2.f *normalVector*NdotL) -lightVector;

  a.r = l->ambient.r * ambient.r;
  a.g = l->ambient.g * ambient.g;
  a.b = l->ambient.b * ambient.b;

  // Diffuse
  d.r = NdotL * l->diffuse.r * diffuse.r * (NdotL > 0);
  d.g = NdotL * l->diffuse.g * diffuse.g * (NdotL > 0);
  d.b = NdotL * l->diffuse.b * diffuse.b * (NdotL > 0);

  // Specular
  RdotV = glm::pow(glm::dot(glm::normalize(reflectVector), viewVector), 100.f);
  s.r = RdotV * l->specular.r * specular.r * (NdotL > 0) *(RdotV>0);
  s.g = RdotV * l->specular.g * specular.g * (NdotL > 0) *(RdotV>0);
  s.b = RdotV * l->specular.b * specular.b * (NdotL > 0) *(RdotV>0);

  total.r = glm::min(1.f, a.r + d.r + s.r);
  total.g = glm::min(1.f, a.g + d.g + s.g);
  total.b = glm::min(1.f, a.b + d.b + s.b);
  total.f = 1.f;
  return total;
}
